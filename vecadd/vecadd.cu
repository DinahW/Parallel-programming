#include "hip/hip_runtime.h"
/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition.
*/

/*Fix the codes that say FIXME*/

#include <stdlib.h>
#include <stdio.h>



// this kernel computes the vector sum c = a + b
// each thread performs one pair-wise addition
__global__ void vector_add(const float *a,
                           const float *b,
                           float *c,
                           const size_t n)

{
  // compute the global element index this thread should process
  /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
  unsigned int index =blockDim.x * blockIdx.x + threadIdx.x + 2; /*FIXME/Fixed #4*/

  // avoid accessing out of bounds elements
  if(index < n)
  {
    // sum elements
    c[index] = a[index] + b[index];
  }
}


int main(void)
{
  // create arrays of 1M elements
  const int num_elements = 1<<20;

  // compute the size of the arrays in bytes
  const int num_bytes = num_elements * sizeof(float);

  // points to host & device arrays
  float *device_array_a = NULL;
  float *device_array_b = NULL;
  float *device_array_c = NULL;
  float *host_array_a   = NULL;
  float *host_array_b   = NULL;
  float *host_array_c   = NULL;

  // malloc the host arrays
  host_array_a = (float*)malloc(num_bytes);
  host_array_b = (float*)malloc(num_bytes);
  host_array_c = (float*)malloc(num_bytes);


  // hipMalloc the device arrays
  hipMalloc((void**)&d_a, size); /*FIXME #1*/);//FIXED
  hipMalloc((void**)&d_b, size); /*FIXME #1*/);//FIXED
  hipMalloc((void**)&d_c, size); /*FIXME #1*/);//FIXED

  // if any memory allocation failed, report an error message
  if(host_array_a == 0 || host_array_b == 0 || host_array_c == 0 ||
     device_array_a == 0 || device_array_b == 0 || device_array_c == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // initialize host_array_a & host_array_b
  for(int i = 0; i < num_elements; ++i)
  {
    // make array a a linear ramp
    host_array_a[i] = (float)i;

    // make array b random
    host_array_b[i] = (float)rand() / RAND_MAX;
  }

  // copy arrays a & b to the device memory space
  /* fix the parameters needed to copy data to the device */
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice); /* FIXME #2*/ );//FIXED
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice); /* FIXME #2*/ );//FIXED

  // compute c = a + b on the device
  const size_t nThreads = 256;
  size_t nBlocks = num_elements / nThreads;

  // deal with a possible partial final block
  if(num_elements % nThreads) ++nBlocks;

  // launch the kernel
  vector_add<<<1,N >>>(d_a, d_b, d_c,N);//FIXED

  // copy the result back to the host memory space
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);//FIXED

  // print out the first 10 results
  for(int i = 0; i < 10; ++i)
  {
    printf("result %d: %1.1f + %7.1f = %7.1f\n", i, host_array_a[i], host_array_b[i], host_array_c[i]);
  }

  // deallocate memory
  free(host_array_a);
  free(host_array_b);
  free(host_array_c);

  hipFree(d_a/*FIXME6*/);//FIXED
  hipFree(d_b/*FIXME6*/);//FIXED
  hipFree(d_c/*FIXME6*/);//FIXED

  return 0;
}
